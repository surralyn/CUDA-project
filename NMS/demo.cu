#include <stdio.h>
#include <time.h>
#include "nms_part_gpu.cu"


int main(){
    FILE *fp = NULL;
    fp = fopen("./data/input.txt", "r");
    int boxes_num;
    float* boxes_host, dt;
    float nms_overlap_thresh;
    fscanf(fp, "%d %f", &boxes_num, &nms_overlap_thresh);
    boxes_host = (float*) malloc(5 * boxes_num * sizeof(float));
    for(int i=0;i<boxes_num;i++){
        for(int j=0;j<5;j++)fscanf(fp, "%f", &boxes_host[5*i + j]);
    }
    hipEvent_t start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    int k=nms_part_gpu(boxes_host, nms_overlap_thresh, boxes_num);
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &dt, start, stop ) );
    printf("\nMy method:\nTime consuming: %fms\n", dt);
    printf("Remaining box num: %d\n", k);

    fclose(fp);
    HANDLE_ERROR( hipEventDestroy( start ) );
 	HANDLE_ERROR( hipEventDestroy( stop ) );
    return 0;
}